#include "hip/hip_runtime.h"
/**
 * @file lidar_simulation.cu
 * @author kekesong (kekesong@tencent.com)
 * @brief 
 * @version 0.1
 * @date 2024-07-12
 * 
 * @copyright Copyright (c) 2024 Tencent Inc.  All rights reserved.
 * 
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <iostream>
#include <string>
#include <vector>
#include "lidar_simulation.h"

using namespace cuda_lidar;
#ifndef CLAMP
#  define CLAMP(x, low, high) (((x) > (high)) ? (high) : (((x) < (low)) ? (low) : (x)))
#endif
#ifndef MAX
#  define MAX(a, b) (((a) > (b)) ? (a) : (b))
#endif
#ifndef MIN
#  define MIN(a, b) (((a) < (b)) ? (a) : (b))
#endif

__device__ float rand_gen(const float d) {
  uint32_t status = d * 123863.f;
  status = status * 21401334u + 2531013245u;
  status = status & 0xffffu;
  return 1.f * (status % 65536) / 32768.f - 1.f;
}

__device__ float lidar_reflection(const float *refmap, uint32_t tag, const float d) {
  float minr = refmap[tag * 3];
  float maxr = refmap[tag * 3 + 1];
  float deta = refmap[tag * 3 + 2];
  float u = 0.5f + deta * rand_gen(d);

  return minr + (maxr - minr) * CLAMP(u, 0.f, 1.f);
}

__device__ float factor_rain(const float d, const float f_rainfall) { return pow(f_rainfall, d); }

__device__ float factor_fog(const float d, const float f_fog) { return factor_rain(d, f_fog); }

__device__ void gausswhite_noise(float &d, const const float f_accuracy, const float f_noise_dev) {
  d += f_accuracy * rand_gen(d) * f_noise_dev;
}

__device__ void snow_noise(float &d, const float f_snowfall_range, const float f_snowfall_prob) {
  float p = rand_gen(d);
  if (p < f_snowfall_prob) return;
  float nd = f_snowfall_range * (rand_gen(d) * 0.5f + 0.5f) + 0.2;
  d = MIN(nd, d);
}

__device__ bool lidar_simulation(const float *refmap, const float cosita, const uint32_t tag, const float t_intensity,
                                 const float f_noise_dev, const float f_accuracy, const float f_rainfall,
                                 const float f_snowfall_range, const float f_snowfall_prob, const float f_fog, float &d,
                                 float &ref) {
  ref = lidar_reflection(refmap, tag, d);
  float v = CLAMP(cosita + 0.2f * (rand_gen(d * cosita) + 0.5f), 0.f, 1.f) * ref / (d * d);
  if (f_rainfall < 1.f) {
    // road water
    if (tag >= 9 && tag <= 19) {
      v *= exp((f_rainfall - 1) * 1024);
    }
    v *= factor_rain(d, f_rainfall);
  }
  if (f_fog < 1.f) {
    v *= factor_fog(d, f_fog);
  }
  if (v < t_intensity) {
    d = 0;
    return false;
  }
  gausswhite_noise(d, f_accuracy, f_noise_dev);
  if (f_snowfall_prob < 1.f) {
    snow_noise(d, f_snowfall_range, f_snowfall_prob);
  }
  return d >= 0.01f;
}

__global__ void lidar_processing(uint8_t **const depthImg, const float *refmap, const uint32_t *tagmap,
                                 const uint8_t *camidx, const uint32_t *camuv, const float *yawpitch,
                                 const float *rtmat, const uint32_t rhn, const float t_intensity,
                                 const float f_noise_dev, const float f_accuracy, const float f_rainfall,
                                 const float f_snowfall_range, const float f_snowfall_prob, const float f_fog,
                                 uint32_t idx, uint32_t N, float *out) {
  uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  if (x < N) {
    uint32_t i = (idx + x) % (rhn);
    int pici = camidx[i];
    if (pici == 255) {
      return;
    }
    if (!depthImg[pici]) {
      return;
    }
    uint32_t xy = camuv[i];
    const uint8_t *color00 = &(depthImg[pici][xy * 4]);
    float distance = ((float)color00[2] * 256.f + (float)color00[1]) * 0.005f;
    if (distance > 0.01f && distance < 327.f) {
      float norinter = (float)color00[0] * 0.00390625f;
      float *ret = out + x * 8;
      ret[3] = distance;
      ret[7] = norinter;
      uint32_t tag = color00[3];
      lidar_simulation(refmap, norinter, tag, t_intensity, f_noise_dev, f_accuracy, f_rainfall, f_snowfall_range,
                       f_snowfall_prob, f_fog, ret[3], ret[4]);
      if (ret[3] > 0.01f && ret[3] < 327.f) {
        *(unsigned int *)(ret + 5) = tagmap[tag * 2];
        *(unsigned int *)(ret + 6) = tagmap[tag * 2 + 1];
        const float *yp = yawpitch + i * 2;
        ret[2] = distance * sin(yp[1]);
        ret[1] = distance * cos(yp[1]) * sin(yp[0]);
        ret[0] = distance * cos(yp[1]) * cos(yp[0]);
        if (rtmat) {
          float nx = rtmat[0] * ret[0] + rtmat[4] * ret[1] + rtmat[8] * ret[2] + rtmat[12];
          float ny = rtmat[1] * ret[0] + rtmat[5] * ret[1] + rtmat[9] * ret[2] + rtmat[13];
          float nz = rtmat[2] * ret[0] + rtmat[6] * ret[1] + rtmat[10] * ret[2] + rtmat[14];
          float nw = rtmat[3] + rtmat[7] + rtmat[11] + rtmat[15];
          ret[0] = nx / nw;
          ret[1] = ny / nw;
          ret[2] = nz / nw;
        }
      }
    }
  }
}

LidarSimulation::LidarSimulation() {}
LidarSimulation::~LidarSimulation() {}

bool LidarSimulation::simulation(uint32_t bg_idx, uint32_t N, float *out, hipStream_t stream) {
  /*if (hipMemsetAsync(out, 0, buf.hn * buf.rn * sizeof(lidarsim_point), stream))
  {
          std::cout << "cannot set out buffer\n";
          return false;
  }*/
  // post_processing
  auto threads_per_block = 128;
  auto blocks = dim3((N + threads_per_block - 1) / threads_per_block);
  auto threads = dim3(threads_per_block);

  lidar_processing<<<blocks, threads, 0, stream>>>(depthImg, refmap, tagmap, camidx, camuv, yawpitch, rtmat, rhn,
                                                   t_intensity, f_noise_dev, f_accuracy, f_rainfall, f_snowfall_range,
                                                   f_snowfall_prob, f_fog, bg_idx, N, out);
  auto err = hipGetLastError();
  if (err) {
    printf("lidar sim error: %d %s", err, hipGetErrorString(err));
    return false;
  }
  return true;
}
