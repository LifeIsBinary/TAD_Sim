#include "hip/hip_runtime.h"
/**
 * @file tadsim_trt_engine.cu
 * @author kekesong (kekesong@tencent.com)
 * @brief
 * @version 0.1
 * @date 2024-07-12
 *
 * @copyright Copyright (c) 2024 Tencent Inc.  All rights reserved.
 *
 */
#include <NvInfer.h>
#include <NvInferPlugin.h>
#include <NvInferRuntime.h>
#include <NvOnnxParser.h>
#include <hip/hip_runtime_api.h>
#include <npp.h>
#include <fstream>
#include <iostream>
#include <sstream>

#include "buffers.h"
#include "common.h"
#include "logger.h"
#include "tadsim_trt_engine.h"
// #include "parserOnnxConfig.h"

using samplesCommon::SampleUniquePtr;

#define CLAMP(x, low, high) (((x) > (high)) ? (high) : (((x) < (low)) ? (low) : (x)))
__global__ void pre_processing(float* plane, uchar* const bgra, const int N) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  if (x < N) {
    plane[x] = 0.0078431372549f * ((float)bgra[x * 4 + 2] - 127.5f);
    plane[x + N] = 0.0078431372549f * ((float)bgra[x * 4 + 1] - 127.5f);
    plane[x + N + N] = 0.0078431372549f * ((float)bgra[x * 4 + 0] - 127.5f);
  }
}
__global__ void post_processing(uchar* bgra, float* const plane, const int N) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  if (x < N) {
    bgra[x * 4] = 127.5f * (CLAMP(plane[x + N + N], -1.0f, 1.0f) + 1.0f);
    bgra[x * 4 + 1] = 127.5f * (CLAMP(plane[x + N], -1.0f, 1.0f) + 1.0f);
    bgra[x * 4 + 2] = 127.5f * (CLAMP(plane[x], -1.0f, 1.0f) + 1.0f);
  }
}

TADSimTRT::TADSimTRT(const TADOnnxParams& params, hipStream_t cudastream, int gpu_id) : mParams(params) {
  hipSetDevice(gpu_id);
  syncstream = cudastream;

  std::cout << mParams.width << std::endl;
  std::cout << mParams.height << std::endl;
  std::cout << mParams.onnxmodel << std::endl;
  std::cout << mParams.cachetrt << std::endl;
  printf("start to build engine\n");
  build();
  printf("findish building engine\n");
}
TADSimTRT::~TADSimTRT() {
  if (mBuffers) delete mBuffers;
  if (mContext) mContext->destroy();
  if (mEngine) mEngine->destroy();
}

bool TADSimTRT::build() {
  std::stringstream gieModelStdStream;
  gieModelStdStream.seekg(0, gieModelStdStream.beg);

  std::ifstream trt_cache(mParams.cachetrt, std::ios::binary);
  if (!trt_cache) {
    if (!ONNX2Trt()) {
      printf("faild to init engine.\n");
      return false;
    } else {
      printf("succ to init engine\n");
    }
    trt_cache.close();
    nvinfer1::IHostMemory* gieModelStream = mEngine->serialize();
    if (!gieModelStream) {
      std::cout << "failed to serialize CUDA engine" << std::endl;
      return false;
    }
    std::ofstream gieModelStdStream(mParams.cachetrt, std::ios::binary);
    std::cout << "model size = " << gieModelStream->size() << std::endl;
    gieModelStdStream.write((const char*)gieModelStream->data(), gieModelStream->size());
    gieModelStream->destroy();
  } else {
    printf("load trt cache\n");
    IRuntime* runtime = createInferRuntime(common::tensorrt::myLogger);
    if (!runtime) {
      return false;
    }
    gieModelStdStream << trt_cache.rdbuf();
    trt_cache.close();
    gieModelStdStream.seekg(0, std::ios::end);
    std::streamsize modelSize = gieModelStdStream.tellg();
    gieModelStdStream.seekg(0, std::ios::beg);
    void* modelMem = malloc(modelSize + 1024);
    gieModelStdStream.read((char*)modelMem, modelSize);

    // This is very important otherwise rt is not running
    initLibNvInferPlugins(&common::tensorrt::myLogger, "");
    mEngine = runtime->deserializeCudaEngine(modelMem, modelSize), samplesCommon::InferDeleter();
    runtime->destroy();
    free(modelMem);

    for (int b = 0; b < mEngine->getNbBindings(); b++) {
      auto dims = mEngine->getBindingDimensions(b);
      std::cout << dims.d[0] << " " << dims.d[1] << " " << dims.d[2] << " " << dims.d[3] << std::endl;

      if (mEngine->bindingIsInput(b)) {
        mInfo.height_in = dims.d[2];
        mInfo.width_in = dims.d[3];
      } else {
        mInfo.height_out = dims.d[2];
        mInfo.width_out = dims.d[3];
      }
    }
  }

  // Create RAII buffer manager object
  auto now = std::chrono::steady_clock::now();
  mBuffers = new samplesCommon::BufferManager(mEngine);
  mContext = mEngine->createExecutionContext();
  if (!mContext) {
    printf("cannot create mContext\n");
    return false;
  }
  inputdev = static_cast<float*>(mBuffers->getDeviceBuffer("input"));
  outputdev = static_cast<float*>(mBuffers->getDeviceBuffer("output"));
  auto n2 = std::chrono::steady_clock::now();
  auto tspan = std::chrono::duration_cast<std::chrono::milliseconds>(n2 - now);
  std::cout << "cast " << tspan.count() << std::endl;

  return true;
}

bool TADSimTRT::ONNX2Trt() {
  auto builder = SampleUniquePtr<nvinfer1::IBuilder>(nvinfer1::createInferBuilder(common::tensorrt::myLogger));
  if (!builder) {
    printf("createInferBuilder falid\n");
    return false;
  }
  const auto explicitBatch = 1U << static_cast<uint32_t>(NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);
  auto network = SampleUniquePtr<nvinfer1::INetworkDefinition>(builder->createNetworkV2(explicitBatch));
  if (!network) {
    printf("createNetworkV2 falid\n");
    return false;
  }
  auto config = SampleUniquePtr<nvinfer1::IBuilderConfig>(builder->createBuilderConfig());
  if (!config) {
    printf("createBuilderConfig falid\n");
    return false;
  }
  auto parser =
      SampleUniquePtr<nvonnxparser::IParser>(nvonnxparser::createParser(*network, common::tensorrt::myLogger));
  if (!parser) {
    printf("createParser falid\n");
    return false;
  }
  auto profileCalib = builder->createOptimizationProfile();
  const auto inputName = "input";  // mParams.inputTensorNames[0];
  const int calibBatchSize{1};
  // We do not need to check the return of setDimension and setCalibrationProfile here as all dims are explicitly set
  profileCalib->setDimensions(inputName, OptProfileSelector::kMIN,
                              Dims4{calibBatchSize, 3, mParams.height, mParams.width});
  profileCalib->setDimensions(inputName, OptProfileSelector::kOPT,
                              Dims4{calibBatchSize, 3, mParams.height, mParams.width});
  profileCalib->setDimensions(inputName, OptProfileSelector::kMAX,
                              Dims4{calibBatchSize, 3, mParams.height, mParams.width});
  config->addOptimizationProfile(profileCalib);

  // constructNetwork
  std::cout << mParams.onnxmodel << std::endl;
  // auto parsed = parser->parseFromFile(mParams.onnxFileName.c_str(), 0);
  auto parsed = parser->parseFromFile(mParams.onnxmodel, 0);
  if (!parsed) {
    printf("constructNetwork falid\n");
    return false;
  }
  config->setMaxWorkspaceSize(16_MiB);
  if (mParams.fp16) {
    config->setFlag(BuilderFlag::kFP16);
  }
  if (mParams.int8) {
    config->setFlag(BuilderFlag::kINT8);
    samplesCommon::setAllDynamicRanges(network.get(), 127.0f, 127.0f);
  }
  samplesCommon::enableDLA(builder.get(), config.get(), mParams.dlaCore);

  // CUDA stream used for profiling by the builder.
  // if(cudastream == NULL)
  //{
  //   cudastream = new hipStream_t;
  //}
  auto profileStream = samplesCommon::makeCudaStream();
  if (!profileStream) {
    printf("makeCudaStream falid");
    return false;
  }
  config->setProfileStream(*profileStream);

  SampleUniquePtr<IHostMemory> plan{builder->buildSerializedNetwork(*network, *config)};
  if (!plan) {
    printf("buildSerializedNetwork falid");
    return false;
  }

  SampleUniquePtr<IRuntime> runtime{createInferRuntime(common::tensorrt::myLogger)};
  if (!runtime) {
    printf("createInferRuntime falid");
    return false;
  }

  mEngine = runtime->deserializeCudaEngine(plan->data(), plan->size()), samplesCommon::InferDeleter();
  if (!mEngine) {
    printf("deserializeCudaEngine falid");
    return false;
  }

  // ASSERT(network->getNbInputs() == 1);
  auto mInputDims = network->getInput(0)->getDimensions();
  mInfo.height_in = mInputDims.d[2];
  mInfo.width_in = mInputDims.d[3];
  // ASSERT(mInputDims.nbDims == 4);

  // ASSERT(network->getNbOutputs() == 1);
  auto mOutputDims = network->getOutput(0)->getDimensions();
  mInfo.height_out = mOutputDims.d[2];
  mInfo.width_out = mOutputDims.d[3];
  //// Caching tensorrt engine file
  // network->destroy();
  // parser->destroy();
  return true;
}

bool TADSimTRT::infer(uchar* input) {
  // post_processing
  auto threads_per_block = 128;
  auto blocks = dim3((mParams.width * mParams.height + threads_per_block - 1) / threads_per_block);
  auto threads = dim3(threads_per_block);

// #define CASTTIME
#ifdef CASTTIME
  hipEvent_t start, stop;
  float time = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
#endif
  pre_processing<<<blocks, threads, 0, syncstream>>>(inputdev, input, mParams.width * mParams.height);
#ifdef CASTTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "cast 1: " << time << std::endl;
#endif

  // rt

  float* bindings[] = {inputdev, outputdev};
#ifdef CASTTIME
  hipEventRecord(start);
#endif
  if (!mContext->enqueueV2((void**)bindings, syncstream, nullptr)) {
    printf("faild enqueueV2\n");
    return false;
  }
#ifdef CASTTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "cast 2: " << time << std::endl;
#endif

#ifdef CASTTIME
  hipEventRecord(start);
#endif
  post_processing<<<blocks, threads, 0, syncstream>>>(input, outputdev, mParams.width * mParams.height);

#ifdef CASTTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "cast 3: " << time << std::endl;
#endif

  hipStreamSynchronize(syncstream);
  return true;
}
