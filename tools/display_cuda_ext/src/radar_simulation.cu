#include "hip/hip_runtime.h"
/**
 * @file radar_simulation.cu
 * @author kekesong (kekesong@tencent.com)
 * @brief 
 * @version 0.1
 * @date 2024-07-12
 * 
 * @copyright Copyright (c) 2024 Tencent Inc.  All rights reserved.
 * 
 */
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <complex>
#include <iostream>
#include <string>
#include <vector>
#include "cuComplexEx.h"
#include "radar_simulation.h"

// CUDA API error checking
#ifndef PI
#  define PI 3.1415926F
#endif
#ifndef MAX
#  define MAX(a, b) (((a) > (b)) ? (a) : (b))
#endif
#ifndef MIN
#  define MIN(a, b) (((a) < (b)) ? (a) : (b))
#endif
#ifndef CLAMP
#  define CLAMP(x, low, high) (((x) > (high)) ? (high) : (((x) < (low)) ? (low) : (x)))
#endif
#ifndef CUDA_RT_CALL
#  define CUDA_RT_CALL(call)                                                    \
    {                                                                           \
      auto status = static_cast<hipError_t>(call);                             \
      if (status != hipSuccess)                                                \
        fprintf(stderr,                                                         \
                "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "      \
                "with "                                                         \
                "%s (%d).\n",                                                   \
                #call, __LINE__, __FILE__, hipGetErrorString(status), status); \
    }
#endif  // CUDA_RT_CALL

// cufft API error chekcing
#ifndef CUFFT_CALL
#  define CUFFT_CALL(call)                                               \
    {                                                                    \
      auto status = static_cast<hipfftResult>(call);                      \
      if (status != HIPFFT_SUCCESS)                                       \
        fprintf(stderr,                                                  \
                "ERROR: CUFFT call \"%s\" in line %d of file %s failed " \
                "with "                                                  \
                "code (%d).\n",                                          \
                #call, __LINE__, __FILE__, status);                      \
    }
#endif  // CUFFT_CALL
using namespace cuda_radar;

__device__ __host__ float db2w(float db) { return powf(10, db * 0.1f); }

__device__ float rand_r(const float d) {
  return 0;
  uint32_t status = d * 123863.f;
  status = status * 21401334u + 2531013245u;
  status = status & 0xffffu;
  return 1.f * (status % 65536) / 32768.f - 1.f;
}

RadarSimulation::RadarSimulation() {}
RadarSimulation::~RadarSimulation() { Clear(); }
float RadarSimulation::L1(int f0, int weather, float r) {
  const static float weather_L1_24[20] = {0.32, 0.618, 2.09, 4.04, 7.99, 0.34, 0.48, 0.64, 0.38,
                                          0.8,  1.28,  0.33, 0.32, 0.32, 0.32, 0.32, 0.32};
  const static float weather_L1_77[20] = {0.62, 2.28, 6.66, 11.02, 18.52, 0.68, 0.94, 1.26,
                                          0.74, 1.58, 2.54, 0.64,  0.62,  0.62, 0.62, 0.62};
  weather = std::min(11, std::max(0, weather));
  if (f0 == 24) {
    return weather_L1_24[weather] * r * 0.001f;
  } else {
    return weather_L1_77[weather] * r * 0.001f;
  }
  return 0;
}

void RadarSimulation::Init() {
  lambda = 0.3f / FC;         // 波长 m
  rangeRes = 0.3f / (2 * B);  // 距离分辨率 m    B = c/(2*rangeRes)
  Tc = B / S;                 // Chirp上升时间 = Nr / Fs = aaa*2*maxR/c
  maxR = Ns * rangeRes;       // 最大需要探测距离 m
  Fs = Ns / Tc;               // 采样频率 Hz  =Nr/Tc
  maxV = lambda / (4 * Tc);   // 最大探测速度 m/s  230*1000/3600
  vres = maxV * 2 / Nd;       // 速度分辨率 m/s

  L1_dB = L1((int)std::round(FC), weather, 1.0f);
  {
    double k = 1.38e-23;
    double Pr = Pt_dBm * Gt_dBi * Gr_dBi * lambda * lambda * Tc;
    double Pn = 64 * PI * PI * PI * k * Ts_K * db2w(Fn_dB) * db2w(L0_dB);
    if (Pr > 0) {
      er_power = Pr / Pn;
    }
  }

  // Create the plans
  size_t workspace_size;
  CUFFT_CALL(hipfftCreate(&plan_r2c));
  CUFFT_CALL(hipfftSetStream(plan_r2c, stream));
  CUFFT_CALL(hipfftMakePlan3d(plan_r2c, Ne, Na, Ns, HIPFFT_R2C, &workspace_size));
  CUDA_RT_CALL(hipMalloc(&iFx, Na * Ne * Ns * sizeof(float)));
}

void RadarSimulation::Clear() {
  if (iFx) CUDA_RT_CALL(hipFree(iFx));
  if (plan_r2c) CUFFT_CALL(hipfftDestroy(plan_r2c));

  iFx = 0;
  plan_r2c = 0;
}

__global__ void signal_sim(uint8_t* rayImg, uint32_t N, float res_range, float res_veolity, float res_aziN,
                           float res_eleN, float rayArea, float lambda, float S, float Er, float L1_dB, float* iFx)

{
  int bid = blockIdx.x * blockDim.x;
  int rid = bid + threadIdx.x;
  if (rid >= N) {
    return;
  }
  uint8_t* rawray = rayImg + rid * 12;

  // get ray info
  float distance = ((float)rawray[2] * 256.f + (float)rawray[1]) * res_range;
  float azith = asin((float)rawray[0] / 127.0f - 1);
  float pitch = asin((float)rawray[3] / 127.0f - 1);
  float3 pos = make_float3(cos(pitch) * cos(azith), cos(pitch) * sin(azith), sin(pitch)) * distance;

  float3 polVec{(float)rawray[6] / 127.0f - 1, (float)rawray[5] / 127.0f - 1, (float)rawray[4] / 127.0f - 1};
  float3 Direction{(float)rawray[10] / 127.0f - 1, (float)rawray[9] / 127.0f - 1, 0};
  Direction.z = -sqrt(MAX(0.000001f, 1 - Direction.x * Direction.x - Direction.y * Direction.y));
  int tag = rawray[7];
  if (tag & 128) {
    Direction.z *= -1;
  }

  // rcs
  const float waveNum = 2 * PI / lambda;
  float3 dirP{0, -1, 0};
  float3 dirT{0, 0, -1};
  float3 vecK{-waveNum, 0, 0};

  hipComplex kr{0, waveNum * distance};
  kr = cuCexpf(kr);
  cuComplexVec3 apE = cuComplexVec3(polVec) * kr;
  cuComplexVec3 apH = -apE.cross(Direction);

  hipComplex BU = (-(apE.cross(-dirP) + apH.cross(dirT))) * Direction;
  hipComplex BR = (-(apE.cross(dirT) + apH.cross(dirP))) * Direction;

  hipComplex factor = make_hipComplex(0, waveNum * rayArea / 4.0 / PI) * cuCexpf(make_hipComplex(0, -vecK * pos));
  BU *= factor;
  BR *= factor;

  extern __shared__ float A[];

  float Am2 = Er / db2w(L1_dB * distance);
  hipComplex AU = BU;
  hipComplex AR = BR;

  float rcs = 4.0 * PI * (pow(hipCabsf(AU), 2) + pow(hipCabsf(AR), 2));
  iFx[rid] = 10. * log10(rcs);
}

bool RadarSimulation::simulation(float* out) {
  auto threads_per_block = 128;
  auto blocks = dim3((N + threads_per_block - 1) / threads_per_block);
  auto threads = dim3(threads_per_block);
#ifdef CASTTIME
  hipEvent_t start, stop;
  float time = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  hipEventRecord(start);
#endif

  signal_sim<<<blocks, threads, 0, stream>>>(rayImg, N, rangeRes, vres, 0, 0, 1, lambda, S, er_power, L1_dB, out);

#ifdef CASTTIME
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  std::cout << "SignalFX cast : " << time << std::endl;
#endif
  auto err = hipGetLastError();
  if (err) {
    printf("lidar sim error: %d %s", err, hipGetErrorString(err));
    return false;
  }
  return true;
}

//__device__ float distance(float x1, float y1, float z1, float x2, float y2, float z2)
//{
//	return sqrt((x1-x2)*(x1-x2)+(y1-y2)*(y1-y2)+(z1-z2)*(z1-z2));
//}
//__device__ float decimal_float_mul(float a, float b)
//{
//	float a1 = floor(a);
//	float a2 = a - a1;
//	float b1 = floor(b);
//	float b2 = b - b1;
//	float c = a1 * b2 + a2 * b1 + b2 * a2;
//	return c - floor(c);
//}
//
//__device__ float radar_rcs(const float *refmap, uint32_t tag, const float d)
//{
//	float minr = refmap[tag * 3];
//	float maxr = refmap[tag * 3 + 1];
//	float deta = refmap[tag * 3 + 2];
//	float u = 0.5f + deta * rand_r(d);
//
//	return minr + (maxr - minr) * CLAMP(u, 0.f, 1.f);
//}
//
//__global__ void signal_sim(uint8_t* pointImg, uint32_t N, uint32_t Ns, uint32_t Na, uint32_t Ne,
//							float lambda, float FC, float S, float er, float L1_dB,
//							float* yawpitch, float* refmap, float* iFx)
//{
//	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
//	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
//	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
//	if(x >= Ns || y >= Na || z >= Ne)
//		return;
//	float Rx = 0;
//	for (uint32_t i = 0; i < N; i++)
//	{
//		const uint8_t *color00 = &(pointImg[i * 4]);
//		uint32_t tag = color00[3];
//		float dis = ((float)color00[2] * 256.f + (float)color00[1]) * 0.005f;
//		const float *yp = yawpitch + i * 2;
//		float tz = dis * sin(yp[1]);
//		float ty = dis * cos(yp[1]) * sin(yp[0]);
//		float tx = dis * cos(yp[1]) * cos(yp[0]);
//		float dis2 = distance(tx,ty,tz,0,-lambda*y/2,lambda*z/2);
//
//		// rcs
//		float rcs = radar_rcs(refmap, tag, dis);
//		// signal
//
//        float Am = er * sqrt(rcs) / pow(dis, 2) / sqrt(db2w(L1_dB*dis));
//
//		// recive
//		const float C = 0.3f;// 光速 e9 m/s
//		float td = -(dis+dis2)/C;//回波时延
//		float ph1 = FC * td;
//		ph1 -= floor(ph1);
//		float ph2 = S * 1e-9f * td * td * 0.5f;
//		ph2 -= floor(ph2);
//        float rx = Am*cos(2*PI* (ph1+ph2));//接收信号波形
//
//        Rx += rx;
//	}
//	iFx[z*Na*Ns+y*Ns+x]=Rx;
//}
// #define CASTTIME
//
// float *RadarSimulation::SignalFX()
//{
//	if (!pointImg || !yawpitch || !refmap || !iFx)
//	{
//		std::cout << "ptr is null";
//		return 0;
//	}
//
//	dim3 dimBlock(16, 4, 2);
//	dim3 dimGrid(std::ceil(Ns * 1.0 / dimBlock.x), std::ceil(Na * 1.0 / dimBlock.y), std::ceil(Ne * 1.0 /
//dimBlock.z));
//
// #ifdef CASTTIME
//	hipEvent_t start, stop;
//	float time = 0;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipEventRecord(start);
//	hipEventRecord(start);
// #endif
//	signal_sim<<<dimGrid, dimBlock, 0, stream>>>(pointImg, N, Ns, Na, Ne,
//													lambda, FC, S,
//er_power, L1_dB, 													yawpitch, refmap, iFx); #ifdef CASTTIME 	hipEventRecord(stop); 	hipEventSynchronize(stop);
//	hipEventElapsedTime(&time, start, stop);
//	std::cout << "SignalFX cast : " << time << std::endl;
// #endif
//
//
//	CUDA_RT_CALL(hipGetLastError());
//    return iFx;
//}
//
// bool RadarSimulation::simulation(float *out)
//{
//	SignalFX();
//	// Execute the plan_r2c
//    CUFFT_CALL(hipfftXtExec(plan_r2c, iFx, iFx, HIPFFT_FORWARD));
//
//
//    return false;
//}
